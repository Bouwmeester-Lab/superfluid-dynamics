#include "hip/hip_runtime.h"
#pragma once
#include "createM.cuh"

/// <summary>
/// Creates the matrix M used in eq. 2.9 from Roberts 1983
/// </summary>
/// <param name="A">Matrix to fill</param>
/// <param name="diag">The precalculated diagonal using the expression for Mkk</param>
/// <param name="n">The size of the matrix (nxn)</param>
/// <returns></returns>
__global__ void createMKernel(double* A, hipfftDoubleComplex* ZPhi, hipfftDoubleComplex* ZPhiPrime, hipfftDoubleComplex* Zpp, double rho, int n)
{
    int k = blockIdx.y * blockDim.y + threadIdx.y; // row
    int j = blockIdx.x * blockDim.x + threadIdx.x; // col

    if (k < n && j < n) {
        int indx = k + j * n; // column major index
        if (k == j) 
        {
            // we are in the diagonal:
            A[indx] = 0.5 * (1 + rho) + 0.25 * (1 - rho) / PI_d * hipCdiv(Zpp[k], ZPhi[k]).y; // imaginary part
        }
        else 
        {
            A[indx] = 0.25 * (1 - rho) / PI_d * (hipCmul(ZPhiPrime[k],  cotangent_complex(cMulScalar(0.5, hipCsub(ZPhi[k], ZPhi[j]))))).y;
        }
    }
}