#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <gtest/gtest.h>


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

TEST(test, TestName)
{
    //This Test will work
    int a[4] = { 1, 2, 3, 4 };
	int b[4] = { 5, 6, 7, 8 };
	int c[4] = { 0, 0, 0, 0 };
    
	int* d_a, * d_b, * d_c;

    hipMalloc(&d_a, 4 * sizeof(int));
	hipMalloc(&d_b, 4 * sizeof(int));
	hipMalloc(&d_c, 4 * sizeof(int));

    hipMemcpy(d_a, a, 4* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, 4 * sizeof(int), hipMemcpyHostToDevice);

	addKernel <<<1, 4 >> > (d_c, d_a, d_b);
	hipMemcpy(c, d_c, 4 * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 4; ++i) {
		EXPECT_EQ(c[i], a[i] + b[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(int argc, char** argv) {
    // Optional: CUDA setup check
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA devices available: " << deviceCount << std::endl;

    // Initialize Google Test
    ::testing::InitGoogleTest(&argc, argv);

    // Run all tests
    return RUN_ALL_TESTS();
}
