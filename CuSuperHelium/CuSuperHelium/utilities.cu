#include "hip/hip_runtime.h"
#pragma once
#include "utilities.cuh"

__global__ void complex_pointwise_mul(
    const hipfftDoubleComplex* a,
    const hipfftDoubleComplex* b,
    hipfftDoubleComplex* result,
    const int n
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        hipfftDoubleComplex x = a[i];
        hipfftDoubleComplex y = b[i];
        result[i].x = x.x * y.x - x.y * y.y;  // real
        result[i].y = x.x * y.y + x.y * y.x;  // imag
    }
}

__global__ void vector_subtract_complex_real(const hipfftDoubleComplex* a, const double* b, hipfftDoubleComplex* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i].x = a[i].x - b[i]; // modifies only the real part.
    }
}

__global__ void vector_scalar_add_complex_real(const hipfftDoubleComplex* a, const double b, hipfftDoubleComplex* out, int n, int start)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[start + i].x = a[start + i].x + b; // modifies only the real part.
    }
}
/// <summary>
/// Calculates the cotangent using cot(z) = cos(z)/sin(z).
/// </summary>
/// <param name="a"></param>
/// <param name="out"></param>
/// <param name="n"></param>
__global__ void cotangent_complex(const hipfftDoubleComplex* a, hipfftDoubleComplex* out, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        hipfftDoubleComplex cs;
        hipfftDoubleComplex cc;

        cos(a[i], cc);
        sin(a[i], cs);

        auto z = hipCdiv(cc, cs);

        out[i].x = z.x;
        out[i].y = z.y;
    }
}

__global__ void real_to_complex(const double* x, hipDoubleComplex* x_c, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
        x_c[idx] = make_hipDoubleComplex(x[idx], 0.0);
}

__device__ static hipfftDoubleComplex cotangent_complex(hipfftDoubleComplex a)
{
    
        hipfftDoubleComplex cs;
        hipfftDoubleComplex cc;

        cos(a, cc);
        sin(a, cs);

        return hipCdiv(cc, cs);   
}

__device__ static void cos(hipfftDoubleComplex z, hipfftDoubleComplex& out)
{
    out.x = cos(z.x) * cosh(z.y);
    out.y = -sin(z.x) * sinh(z.y);
}

__device__ static void sin(hipfftDoubleComplex z, hipfftDoubleComplex& zout) {
    zout.x = sinh(z.x) * cos(z.y);
    zout.y = cosh(z.x) * sin(z.y);
}

__device__ hipfftDoubleComplex fromReal(double a)
{
    hipfftDoubleComplex out;
    out.x = a;
    out.y = 0.0;
    return out;
}

hipfftDoubleComplex cMulScalar(double a, hipfftDoubleComplex z)
{
    hipfftDoubleComplex out(z);

    out.x = a * out.x;
    out.y = a * out.y;

    return out;
}

void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCusolver(hipsolverStatus_t status) {
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "cuSolver Error" << std::endl;
        exit(EXIT_FAILURE);
    }
}
