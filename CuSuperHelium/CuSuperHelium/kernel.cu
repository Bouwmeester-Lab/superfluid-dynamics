#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Derivatives.hpp"
#include <hipfft/hipfft.h>
#include "array"
#include "constants.cuh"

hipError_t setDevice();
hipError_t fftDerivative();

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 8;
    const int a[arraySize] = { 1, 2, 3, 4, 5, 6, 7 };
    const int b[arraySize] = { 10, 20, 30, 40, 50, 60, 70 };

    const double X0[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7 };
    const double Y0[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7 };

    int c[arraySize] = { 0 };

    // test fft derivative
    hipError_t cudaStatus = fftDerivative();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t fftDerivative()
{
    hipError_t cudaStatus;
    cudaStatus = setDevice();
    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    

    const int N = 16;

    std::array<hipfftDoubleComplex, N> y;
    std::array<hipfftDoubleComplex, N> yp;
    hipfftDoubleComplex* devY;
    hipfftDoubleComplex* devYp;

    for (int i = 0; i < N; i++) 
    {
        y[i].x = sin(2.0 * 2.0 * i * PI_d / (1.0*N));
        y[i].y = 0;
    }

    FftDerivative<N, 1> derivativeFft;

    cudaStatus = derivativeFft.initialize();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "failed to initialize the fft derivatives! ");
        return cudaStatus;
    }
    
    cudaStatus = hipMalloc(&devY, sizeof(hipfftDoubleComplex) * N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "failed to allocate devY to gpu! ");
        return cudaStatus;
    }
    // copy y to the gpu
    cudaStatus = hipMemcpy(devY, y.data(), sizeof(hipfftDoubleComplex) * N, hipMemcpyHostToDevice);
    cudaStatus = hipMalloc(&devYp, sizeof(hipfftDoubleComplex) * N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "failed to copy to gpu! ");
        return cudaStatus;
    }

    derivativeFft.exec(devY, devYp);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(yp.data(), devYp, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }
    printf("end");
    printf("d/dj y {%d, %d, %d, ... } = {%d,%d,%d, ...}\n",
        y[0].x, y[1].x, y[2].x, yp[0].x, yp[1].x, yp[2].x);
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t setDevice()
{
    
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    return cudaStatus;
    
    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
}
