#include "hip/hip_runtime.h"
#pragma once
#include "utilities.cuh"

__global__ void complex_pointwise_mul(
    const hipfftDoubleComplex* a,
    const hipfftDoubleComplex* b,
    hipfftDoubleComplex* result,
    const int n
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        hipfftDoubleComplex x = a[i];
        hipfftDoubleComplex y = b[i];
        result[i].x = x.x * y.x - x.y * y.y;  // real
        result[i].y = x.x * y.y + x.y * y.x;  // imag
    }
}