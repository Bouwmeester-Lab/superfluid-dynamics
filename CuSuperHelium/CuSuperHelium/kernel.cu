#include "hip/hip_runtime.h"
﻿
//#define DEBUG_DERIVATIVES
//#define DEBUG_RUNGE_KUTTA
//#define DEBUG_DERIVATIVES_3
//#define DEBUG_VELOCITIES
#include "ProblemProperties.hpp"
#include "array"
#include "vector"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <hipfft/hipfft.h>

#include "constants.cuh"
#include <complex>

#include "WaterBoundaryIntegralCalculator.cuh"
#include "SimpleEuler.cuh"
#include "AutonomousRungeKuttaStepper.cuh"
#include "ValueLogger.h"
#include "SolitonPeak.h"

#include <format>
//#include "math.h"
//#include "complex.h"
#include "matplotlibcpp.h"
#include "SimulationRunner.cuh"
#include <chrono>
namespace plt = matplotlibcpp;

#define j_complex std::complex<double>(0, 1)
hipError_t setDevice();


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

double X(double j, double h, double omega, double t) {
    return j - h * std::sin((j - omega * t));
}

double Y(double j, double h, double omega, double t) {
    return h * PeriodicFunctions::gaussian::gaussian_periodic(j) * std::sin(j);  // std::cos((j - omega * t));
}

double Phi(double j, double h, double omega, double t, double rho) {
    return h * (1 + rho) * omega * PeriodicFunctions::bimodal::bimodal(j);// std::sech2_periodic((j - omega * t));
}

int main() 
{
	auto time0 = std::chrono::high_resolution_clock::now();

	ProblemProperties problemProperties;
    problemProperties.rho = 0;
	problemProperties.kappa = 0;
    problemProperties.U = 0;
    problemProperties.tolerance = 1e-8;
	problemProperties.maxIterations = 200;
    
    int frames = 22;
    double omega = 1;
    double t0 = 0;
	double finalTime = 10e-6; // 15 ms
    
    double H0 = 15e-9; // 15 nm
    double g = 3 * 2.6e-24 / std::pow(H0, 4); //
	double L0 = 1000e-6/(2.0*PI_d); // 1mm

    double _t0 = std::sqrt(L0 / g);

    problemProperties.depth = H0 / L0;
    double h = 0.1 * problemProperties.depth;

	problemProperties.initial_amplitude = h;
	problemProperties.y_min = -h - 0.0001 * problemProperties.depth; // -0.5 * H0
	problemProperties.y_max = h +  0.005 * problemProperties.depth; // 0.5 * H0
	printf("Simulating with depth (h_0) %.10e, h %.10e, omega %f, t0 %.10e, L0 %.10e\n", problemProperties.depth, h, omega, _t0, L0);
	printf("g %.10e, H0 %.10e, L0 %.10e\n", g, H0, L0);

    const int N = 512;//512;
    
	const double stepSize = 0.1;
    const int steps = (finalTime / _t0) / stepSize;
	const int loggingSteps = steps / frames;

    printf("Simulating %i steps representing %.2e s", steps, steps * stepSize * _t0);
    

    std::array<std_complex, N> Z0;
	std::vector<double> X0(N, 0);
    std::vector<double> Y0(N, 0);
    
	std::array<std_complex, N> PhiArr;

    std::vector<double> Phireal(N, 0);
    double j;
	for (int i = 0; i < N; i++) {
		j = 2.0 * PI_d * i / (1.0 * N);
		Z0[i] = std_complex(X(j, h, omega, t0), Y(j, h, omega, t0));
        PhiArr[i] = std_complex(0.0*Phi(j, h, omega, t0, problemProperties.rho), 0.0);
		Phireal[i] = PhiArr[i].real();

		X0[i] = Z0[i].real();
		Y0[i] = Z0[i].imag();
	}

	/*plt::figure();
    plt::plot(X0, Phireal);
	plt::plot(X0, Y0);
    plt::show();*/

    ParticleData particleData;
	particleData.Z = Z0.data();
	particleData.Potential = PhiArr.data();

    
     auto res = runSimulationHelium<N>(steps, stepSize, problemProperties, particleData, loggingSteps, true, false, _t0);

	 auto time1 = std::chrono::high_resolution_clock::now();
	 std::chrono::duration<double, std::milli> elapsed = time1 - time0;

     std::cout << "Elapsed time: " << elapsed.count() << " ms\n";
}