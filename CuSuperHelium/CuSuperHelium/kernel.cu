#include "hip/hip_runtime.h"
﻿
//#define DEBUG_DERIVATIVES
//#define DEBUG_RUNGE_KUTTA
//#define DEBUG_VELOCITIES
#include "ProblemProperties.hpp"
#include "array"
#include "vector"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <hipfft/hipfft.h>

#include "constants.cuh"
#include <complex>

#include "TimeStepManager.cuh"
#include "SimpleEuler.cuh"
#include "RungeKunta.cuh"

#include <format>
//#include "math.h"
//#include "complex.h"
#include "matplotlibcpp.h"
namespace plt = matplotlibcpp;

#define j_complex std::complex<double>(0, 1)
hipError_t setDevice();
hipError_t fftDerivative();


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

double X(double j, double h, double omega, double t) {
    return j - h * std::sin((j - omega * t));
}

double Y(double j, double h, double omega, double t) {
	return h * std::cos((j - omega * t));
}

double Phi(double j, double h, double omega, double t, double rho) {
    return h * ((1 + rho) * omega * std::sin((j - omega * t)));
}

int runTimeStep() 
{

    hipError_t cudaStatus;
    cudaStatus = setDevice();
    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

	ProblemProperties problemProperties;
    problemProperties.rho = 0;
	problemProperties.kappa = 0;
    problemProperties.U = 0;

    const int N = 16;
    const int steps = 10;
	double stepSize = 5e-2;
	WaterBoundaryIntegralCalculator<N> timeStepManager(problemProperties);

	std::array<double, N> j;
    std::vector<double> x0;
	std::vector<double> y0;

    std::array<hipfftDoubleComplex, N> Z0;
	std::array<hipfftDoubleComplex, N> PhiArr;
    std::vector<double> phiPrime;
    std::vector<double> phi0;

	std::array<hipfftDoubleComplex, N> VelocitiesLower;
    std::array<hipfftDoubleComplex, N> VelocitiesUpper;

    std::array<hipfftDoubleComplex, N> ZVect;
    std::array<hipfftDoubleComplex, N> PhiVect;



    std::vector<double> x;
	std::vector<double> y;

	x0.resize(N, 0);
	y0.resize(N, 0);
	phi0.resize(N, 0);
    x.resize(N, 0);
	y.resize(N, 0);
	phiPrime.resize(N, 0);
    double h = 0.1;
    double omega = 1;
    double t0 = 0;
	for (int i = 0; i < N; i++) {
		j[i] = 2.0 * PI_d * i / (1.0 * N);
		Z0[i].x = X(j[i], h, omega, t0);
		x0[i] = Z0[i].x;

		Z0[i].y = Y(j[i], h, omega, t0);
		y0[i] = Z0[i].y;

        PhiArr[i].x = Phi(j[i], h, omega, t0, problemProperties.rho);
		phi0[i] = PhiArr[i].x;

        PhiArr[i].y = 0; // Phi is real.
	}
    plt::figure();
    plt::title("Interface And Potential");
    plt::plot(x0, y0, {{"label", "Interface"}});
	plt::plot(x0, phi0, {{"label", "Potential"}});
    plt::legend();
    //plt::show();
    
	// Initialize the time step manager with the initial conditions.
	hipDoubleComplex* devZ = nullptr;
	hipDoubleComplex* devPhi = nullptr;

    timeStepManager.initialize_device(Z0.data(), PhiArr.data(), devZ, devPhi);
    /*timeStepManager.runTimeStep();
    hipDeviceSynchronize();
	hipMemcpy(phiPrime.data(), timeStepManager.devPhiPrime, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(VelocitiesLower.data(), timeStepManager.devVelocitiesLower, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    printf("\velocities after 1: ");
    for (int i = 0; i < N; i++) {
        printf("{%f, %f} ", VelocitiesLower[i].x, VelocitiesLower[i].y);
        x[i] = ZVect[i].x;
        y[i] = ZVect[i].y;
    }
    plt::figure();
    plt::plot(x0, phi0);
    plt::plot(x0, phiPrime);
    plt::show();*/
    // create Euler stepper
	RungeKuntaStepper<N> rungeKunta(timeStepManager, stepSize);
	// Euler<N> euler(timeStepManager, stepSize);
	/*euler.setDevZ(devZ);
	euler.setDevPhi(devPhi);*/
    rungeKunta.initialize(devZ);
	rungeKunta.setTimeStep(stepSize);
	for (int i = 0; i < steps; i++) {
        // Perform a time step
		rungeKunta.step();
	}
	

    // timeStepManager.runTimeStep();
	hipDeviceSynchronize();

	hipMemcpy(ZVect.data(), devZ, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(PhiVect.data(), devPhi, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(VelocitiesLower.data(), timeStepManager.devVelocitiesLower, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    //hipMemcpy(PhiVect.data(), devPhi, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

	printf("\velocities after 1: ");
	double t = steps * stepSize;
	std::vector<double> x_fin(N, 0);
	std::vector<double> y_fin(N, 0);

	for (int i = 0; i < N; i++) {
		printf("{%f, %f} ", VelocitiesLower[i].x, VelocitiesLower[i].y);
        x[i] = ZVect[i].x;
        y[i] = ZVect[i].y;

		x_fin[i] = X(j[i], h, omega, t);
		y_fin[i] = Y(j[i], h, omega, t);
	}
	printf("\n");
    printf("\nPhi: ");
    for (int i = 0; i < N; i++) {
        printf("{%f, %f} ", PhiVect[i].x, -1 * PhiVect[i].y);
    }
    plt::figure();
    auto title = std::format("Interface And Potential at t={:.4f}", steps * stepSize);
	plt::title(title);

    //plt::plot(x_fin, y_fin, {{"label", "Interface at t=" + std::to_string(t)}});
    // Plot the initial position and the result of the Euler method

	plt::plot(x0, y0, {{"label", "Initial Position"}});
    plt::scatter(x, y);
    plt::legend();
    plt::show();
    

    return 0;
}

int main()
{
    //Py_SetPythonHome(L"C:/ProgramData/anaconda3");

    runTimeStep();
    //const int arraySize = 8;
    //const int a[arraySize] = { 1, 2, 3, 4, 5, 6, 7 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50, 60, 70 };

    //const double X0[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7 };
    //const double Y0[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7 };

    //int c[arraySize] = { 0 };

    //// test fft derivative
    //hipError_t cudaStatus = fftDerivative();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

hipError_t fftDerivative()
{
    hipError_t cudaStatus;
    cudaStatus = setDevice();
    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    

    const int N = 16;

    std::array<hipfftDoubleComplex, N> y;
    std::array<hipfftDoubleComplex, N> yp;
    hipfftDoubleComplex* devY;
    hipfftDoubleComplex* devYp;

    for (int i = 0; i < N; i++) 
    {
        y[i].x = sin(2.0 * 2.0 * i * PI_d / (1.0*N));
        y[i].y = 0;
    }

    FftDerivative<N, 1> derivativeFft;

    cudaStatus = derivativeFft.initialize();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "failed to initialize the fft derivatives! ");
        return cudaStatus;
    }
    
    cudaStatus = hipMalloc(&devY, sizeof(hipfftDoubleComplex) * N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "failed to allocate devY to gpu! ");
        return cudaStatus;
    }
    // copy y to the gpu
    cudaStatus = hipMemcpy(devY, y.data(), sizeof(hipfftDoubleComplex) * N, hipMemcpyHostToDevice);
    cudaStatus = hipMalloc(&devYp, sizeof(hipfftDoubleComplex) * N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "failed to copy to gpu! ");
        return cudaStatus;
    }

    derivativeFft.exec(devY, devYp);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(yp.data(), devYp, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }
    printf("end");
    printf("d/dj y {%d, %d, %d, ... } = {%d,%d,%d, ...}\n",
        y[0].x, y[1].x, y[2].x, yp[0].x, yp[1].x, yp[2].x);
	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t setDevice()
{
    
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    return cudaStatus;
    
    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
}
