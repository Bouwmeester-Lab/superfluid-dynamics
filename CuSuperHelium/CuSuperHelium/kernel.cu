#include "hip/hip_runtime.h"
﻿
//#define DEBUG_DERIVATIVES
//#define DEBUG_RUNGE_KUTTA
//#define DEBUG_VELOCITIES
#include "ProblemProperties.hpp"
#include "array"
#include "vector"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <hipfft/hipfft.h>

#include "constants.cuh"
#include <complex>

#include "WaterBoundaryIntegralCalculator.cuh"
#include "SimpleEuler.cuh"
#include "AutonomousRungeKuttaStepper.cuh"
#include "ValueLogger.h"

#include <format>
//#include "math.h"
//#include "complex.h"
#include "matplotlibcpp.h"
namespace plt = matplotlibcpp;

#define j_complex std::complex<double>(0, 1)
hipError_t setDevice();


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

double X(double j, double h, double omega, double t) {
    return j - h * std::sin((j - omega * t));
}

double Y(double j, double h, double omega, double t) {
	return h * std::cos((j - omega * t));
}

double Phi(double j, double h, double omega, double t, double rho) {
    return h * ((1 + rho) * omega * std::sin((j - omega * t)));
}

int runTimeStep() 
{

    hipError_t cudaStatus;
    cudaStatus = setDevice();
    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

	ProblemProperties problemProperties;
    problemProperties.rho = 0;
	problemProperties.kappa = 0;
    problemProperties.U = 0;
    problemProperties.depth = 0.11;
    double h = 0.1;
    double omega = 1;
    double t0 = 0;

    const int N = 128;
    
	const double stepSize = PI_d/4000;
	const int steps = 0.1 / stepSize;
	const int loggingSteps = steps / 10;
    HeliumBoundaryProblem<N> boundaryProblem(problemProperties);
	BoundaryIntegralCalculator<N> timeStepManager(problemProperties, boundaryProblem);

	std::array<double, N> j;
    std::vector<double> x0;
	std::vector<double> y0;

    std::array<std_complex, N> Z0;
	std::array<std_complex, N> PhiArr;
    std::vector<double> phiPrime;
    std::vector<double> phi0;
    std::vector<std_complex> dPhi(N);

	std::array<std_complex, N> VelocitiesLower;
    std::array<std_complex, N> VelocitiesUpper;

    std::array<std_complex, N> ZVect;
    std::array<std_complex, N> PhiVect;



    std::vector<double> x;
	std::vector<double> y;
	
	ValueLogger kineticEnergyLogger(loggingSteps);
	ValueLogger potentialEnergyLogger(loggingSteps);
	ValueLogger surfaceEnergyLogger(loggingSteps);
	ValueLogger totalEnergyLogger(loggingSteps);
	ValueLogger volumeFluxLogger(loggingSteps);

    std::vector<double> loggedSteps(steps / loggingSteps + 1, 0);
	printf("N = %d, steps = %d, loggingSteps = %d\n", N, steps, loggingSteps);
	printf("loggedSteps size = %d\n", loggedSteps.size());
	

	x0.resize(N, 0);
	y0.resize(N, 0);
	phi0.resize(N, 0);
    x.resize(N, 0);
	y.resize(N, 0);
	phiPrime.resize(N, 0);
    
	for (int i = 0; i < N; i++) {
		j[i] = 2.0 * PI_d * i / (1.0 * N);
		Z0[i] = std_complex(X(j[i], h, omega, t0), Y(j[i], h, omega, t0));
		x0[i] = Z0[i].real();
		y0[i] = Z0[i].imag();

        PhiArr[i] = Phi(j[i], h, omega, t0, problemProperties.rho);
		phi0[i] = PhiArr[i].real();
	}
    plt::figure();
    plt::title("Interface And Potential");
    plt::plot(x0, y0, {{"label", "Interface"}});
	plt::plot(x0, phi0, {{"label", "Potential"}});
    plt::legend();
    //plt::show();
    
	// Initialize the time step manager with the initial conditions.
    timeStepManager.initialize_device(Z0.data(), PhiArr.data());
    
    timeStepManager.runTimeStep();
    hipDeviceSynchronize();
	hipMemcpy(dPhi.data(), timeStepManager.devRhsPhi, N * sizeof(std_complex), hipMemcpyDeviceToHost);
    hipMemcpy(VelocitiesLower.data(), timeStepManager.devVelocitiesLower, N * sizeof(std_complex), hipMemcpyDeviceToHost);
    printf("\velocities after 1: ");
    for (int i = 0; i < N; i++) {
        printf("{%f, %f} ", VelocitiesLower[i].real(), VelocitiesLower[i].imag());
        x[i] = VelocitiesLower[i].real();
        y[i] = VelocitiesLower[i].imag();
        phiPrime[i] = dPhi[i].real();
    }
    plt::figure();
    // plt::plot(x0, phi0);
    plt::title(std::format("Starting RHS using CUDA C++ using N = {}", N));
    plt::plot(x0, x, {{"label", "vx"}});
    plt::plot(x0, y, {{"label", "vy"}});
    plt::plot(x0, phiPrime, { {"label", "dPhi"} });
    plt::legend();

    DataLogger<std_complex, 2 * N> stateLogger;
    stateLogger.setSize(steps / loggingSteps + 1);
    stateLogger.setStep(loggingSteps);
    // plt::show();
    // create Euler stepper
	AutonomousRungeKuttaStepper<std_complex, 2*N> rungeKunta(timeStepManager, stateLogger, stepSize);
	// Euler<N> euler(timeStepManager, stepSize);
	/*euler.setDevZ(devZ);
	euler.setDevPhi(devPhi);*/
    rungeKunta.initialize(timeStepManager.getY0());
	rungeKunta.setTimeStep(stepSize);

	for (int i = 0; i < steps; i++) {
        // Perform a time step
        rungeKunta.runStep(i);
		//hipDeviceSynchronize();
        if (kineticEnergyLogger.shouldLog(i)) {
			kineticEnergyLogger.logValue(boundaryProblem.energyContainer.kineticEnergy->getEnergy());
        }
        if (potentialEnergyLogger.shouldLog(i)) {
			potentialEnergyLogger.logValue(boundaryProblem.energyContainer.potentialEnergy->getEnergy());
        }
		if(volumeFluxLogger.shouldLog(i)) {
            volumeFluxLogger.logValue(timeStepManager.volumeFlux.getEnergy());
        }
		if(totalEnergyLogger.shouldLog(i)) {
            totalEnergyLogger.logValue(kineticEnergyLogger.getLastLoggedValue() + potentialEnergyLogger.getLastLoggedValue());
		}
        if (i % loggingSteps == 0) {
            loggedSteps[i / loggingSteps] = i;
        }
	}
	
	printf("Energy has %i values", kineticEnergyLogger.getLoggedValuesCount());

    // timeStepManager.runTimeStep();
	hipDeviceSynchronize();

	hipMemcpy(ZVect.data(), timeStepManager.getDevZ(), N * sizeof(std_complex), hipMemcpyDeviceToHost);
    hipMemcpy(PhiVect.data(), timeStepManager.getDevPhi(), N * sizeof(std_complex), hipMemcpyDeviceToHost);
    hipMemcpy(VelocitiesLower.data(), timeStepManager.devVelocitiesLower, N * sizeof(std_complex), hipMemcpyDeviceToHost);
    //hipMemcpy(PhiVect.data(), devPhi, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

	printf("\velocities after 1: ");
	double t = steps * stepSize;
	std::vector<double> x_fin(N, 0);
	std::vector<double> y_fin(N, 0);

	for (int i = 0; i < N; i++) {
		printf("{%f, %f} ", VelocitiesLower[i].real(), VelocitiesLower[i].imag());
        x[i] = ZVect[i].real();
        y[i] = ZVect[i].imag();

		x_fin[i] = X(j[i], h, omega, t);
		y_fin[i] = Y(j[i], h, omega, t);

	}
	printf("\n");
    printf("\nPhi: ");
    for (int i = 0; i < N; i++) {
        printf("{%f, %f} ", PhiVect[i].real(), -1 * PhiVect[i].imag());
    }
    plt::figure();
    auto title = std::format("Interface And Potential at t={:.4f}", steps * stepSize);
	plt::title(title);

	auto& timeStepData = stateLogger.getAllData();

    for (int i = 0; i < timeStepData.size(); i++) {
        auto& stepData = timeStepData[i];
        std::vector<double> x_step(N, 0);
        std::vector<double> y_step(N, 0);
        for (int j = 0; j < N; j++) {
            x_step[j] = stepData[j].real();
            y_step[j] = stepData[j].imag();
        }
        plt::plot(x_step, y_step, {{"label", "Interface at t=" + std::to_string(i * stepSize)}});
	}

    //plt::plot(x_fin, y_fin, {{"label", "Interface at t=" + std::to_string(t)}});
    // Plot the initial position and the result of the Euler method

	plt::plot(x0, y0, {{"label", "Initial Position"}});
    plt::plot(x, y);
    plt::legend();

    plt::figure();
	plt::title("Kinetic, Potential and Surface Energy");
	plt::plot(loggedSteps, kineticEnergyLogger.getLoggedValues(), {{"label", "Kinetic Energy"}});
	plt::plot(loggedSteps, potentialEnergyLogger.getLoggedValues(), {{"label", "Potential Energy"}});
	plt::plot(loggedSteps, totalEnergyLogger.getLoggedValues(), {{"label", "Total Energy"}});

	plt::legend();
	plt::xlabel("Time Steps");
	plt::ylabel("Energy");

	plt::figure();
	plt::title("Volume Flux");
	plt::plot(loggedSteps, volumeFluxLogger.getLoggedValues(), {{"label", "Volume Flux"}});
	plt::xlabel("Time Steps");
	plt::ylabel("Volume Flux");
	plt::legend();

    plt::show();
    

    return 0;
}

int main()
{
    //Py_SetPythonHome(L"C:/ProgramData/anaconda3");

    runTimeStep();
    //const int arraySize = 8;
    //const int a[arraySize] = { 1, 2, 3, 4, 5, 6, 7 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50, 60, 70 };

    //const double X0[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7 };
    //const double Y0[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7 };

    //int c[arraySize] = { 0 };

    //// test fft derivative
    //hipError_t cudaStatus = fftDerivative();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t setDevice()
{
    
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    return cudaStatus;
    
    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
}
